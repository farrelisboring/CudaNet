#include "hip/hip_runtime.h"
#include "exports.h"
#include "CudaNet.h"
#include <iostream>
#include <hip/hip_runtime.h>

 __global__ void KernelMultiplyFloat32(const float* __restrict__ a_ptr, const float* __restrict__ b_ptr, float* c_ptr, int a_row, int shared_dimension, int b_col) {
         int row = blockIdx.y * blockDim.y + threadIdx.y;
         int col = blockIdx.x * blockDim.x + threadIdx.x;

         if (row < a_row && col < b_col) {
             float sum = 0.0f;
             for (int r = 0; r < shared_dimension; ++r) {
                 float a = a_ptr[row * shared_dimension + r]; //1d
                 float b = b_ptr[r * b_col + col];// 1d
                 sum += a * b;
             }
             c_ptr[row * b_col + col] = sum;
         }
 }
 
    

extern "C" {


    MY_API void MatrixMultiplyFloat32(const float* __restrict__ a_ptr, const float* __restrict__ b_ptr, float* __restrict__ c_buffer, int a_row, int shared_dimension, int b_col) { 
        
        float* allocated_a, * allocated_b, * allocated_c; 
        size_t a_size = sizeof(float) * (size_t) a_row * (size_t)shared_dimension;
        size_t b_size = sizeof(float) * (size_t) b_col * (size_t)shared_dimension;
        size_t c_size = sizeof(float) * (size_t) a_row * (size_t) b_col;


        hipMalloc((void**)&allocated_a, a_size);
        hipMalloc((void**)&allocated_b, b_size);
        hipMalloc((void**)&allocated_c, c_size);

        hipMemcpy(allocated_b, b_ptr, b_size, hipMemcpyHostToDevice);
        hipMemcpy(allocated_a, a_ptr, a_size, hipMemcpyHostToDevice);


        dim3 block(16, 16);
        dim3 grid((b_col + block.x - 1) / block.x, (a_row + block.y - 1) / block.y);

        KernelMultiplyFloat32 <<< grid, block >>> (allocated_a, allocated_b, allocated_c, a_row, shared_dimension, b_col);

        hipMemcpy(c_buffer, allocated_c, c_size, hipMemcpyDeviceToHost);

        
        hipFree(allocated_a);
        hipFree(allocated_b);
        hipFree(allocated_c);
    }
}

